#include "hip/hip_runtime.h"
#include <sentinel.h>
#include <assert.h>
//
#define HostDir "C:\\T_\\"
#define DeviceDir ":\\"

static __global__ void g_testbed();

int main() {
	float milliseconds = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		goto Error;
	}

	cudaErrorCheck(hipDeviceSetLimit(hipLimitStackSize, 1024 * 5));
	sentinelServerInitialize();

	//PIDTYPE *pids;
	//FDTYPE inPipe, outPipe, errFile;
	//char *pipeArgs = { "C:\\Windows\\System32\\more.com" };
	//int pipeRet = CreatePipeline(1, &pipeArgs, &pids, &inPipe, &outPipe, &errFile);
	//free(pids);
	//goto Error;


	// Launch test
	hipEventRecord(start);
	g_testbed << <1, 1 >> > ();
	hipEventRecord(stop);
	//
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "test failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "test launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Effective: %fn", milliseconds / 1e6);

Error:
	sentinelServerShutdown();

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!\n");
		return 1;
	}

	// finish
	printf("\nPress any key to continue.\n");
	char c; scanf("%c", &c);

	return 0;
}

static __constant__ const char *_quickbrownfox =
"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog."
"The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog. The quick brown fox jumped over the lazy dog.";

//static __device__ char _buf0[] = "The quick brown fox jumped over the lazy dog.";
static __device__ char _buf1[50];

static __global__ void g_testbed() {
}
