#include <sentinel.h>
#include <stdlib.h>
#include <stdio.h>

hipError_t sentinel_test1();

#if _HASPAUSE
#define mainPause(fmt) { printf(fmt"\n"); char c; scanf("%c", &c); }
#else
#define mainPause(fmt) { printf(fmt"\n"); }
#endif

int main(int argc, char **argv) {
	int testId = argv[1] ? atoi(argv[1]) : 11; // 23;

	// Choose which GPU to run on, change this on a multi-GPU system.
	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		goto Error;
	}
	cudaErrorCheck(hipDeviceSetLimit(hipLimitStackSize, 1024 * 5));
	sentinelServerInitialize();

	// Launch test
	switch (testId) {
	case 0: mainPause("Press any key to continue."); break;
	case 11: cudaStatus = sentinel_test1(); break;
	}
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "failed! %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// finish
	mainPause("SUCCESS");

Error:
	sentinelServerShutdown();

	// close
	if (cudaStatus != hipSuccess) {
		// finish
		mainPause("ERROR");
		return 1;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!\n");
		return 1;
	}

	return 0;
}
